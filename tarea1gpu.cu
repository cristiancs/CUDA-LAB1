

#include <hip/hip_runtime.h>
#include<iostream>
#include <fstream>
#include <string>
#include <stdio.h>
#include <stdlib.h>
using namespace std;

__global__ void kernel( float* r_gpu,  float* g_gpu,   float* b_gpu, int N, int n_m) {
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	int i=n_m;
	while(i < N && tId < n_m) {
			r_gpu[tId] += r_gpu[tId+i];
			g_gpu[tId] += g_gpu[tId+i];
			b_gpu[tId] += b_gpu[tId+i];
			i+=n_m;
	}
}

__global__ void kernel2( float* r_gpu,  float* g_gpu,   float* b_gpu, int N, int l) {
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	if(tId < N) {
		r_gpu[tId] = r_gpu[tId]/l;
		g_gpu[tId] = g_gpu[tId]/l;
		b_gpu[tId] = b_gpu[tId]/l;
	}
}


void sumar(float* r1, float* g1, float* b1, float* r, float* g, float* b) {
	//cout << *r << "|" << 1-*r << endl;
	*r1 += *r;
	*g1 += *g;
	*b1 += *b;
}

void promedio(float* r, float* g, float* b, int l) {
	*r = *r/l;
	*g = *g/l;
	*b = *b/l;
}


int main(int argc, char const *argv[]) {
	FILE *pFile;
	int n, m, l;
	float *r, *g, *b; 
	pFile = fopen ("images6.txt","r");
	fscanf(pFile, "%d %d %d", &l, &m, &n);

	r = new float[n*m*l];
	g = new float[n*m*l];
	b = new float[n*m*l];

	int block_size = 256;
	int grid_size = (int) ceil((float) n*m / block_size);



	float* r_gpu, *g_gpu, *b_gpu;

	hipMalloc(&r_gpu, sizeof(float) * n * m * l);
	hipMalloc(&g_gpu, sizeof(float) * n * m * l);
	hipMalloc(&b_gpu, sizeof(float) * n * m * l);

	for (int j = 0; j < l; ++j){
		for (int i = 0; i < n*m; ++i) {
			fscanf (pFile, "%f", &r[i+(j*n*m)]);
		}

		for (int i = 0; i < n*m; ++i) {
			fscanf (pFile, "%f", &g[i+(j*n*m)]);
		}

		for (int i = 0; i < n*m; ++i) {
			fscanf (pFile, "%f", &b[i+(j*n*m)]);
		}
	}



	fclose (pFile);

	hipMemcpy(r_gpu, r, sizeof(float) * n * m * l, hipMemcpyHostToDevice);
	hipMemcpy(g_gpu, g, sizeof(float) * n * m * l, hipMemcpyHostToDevice);
	hipMemcpy(b_gpu, b, sizeof(float) * n * m * l, hipMemcpyHostToDevice);

	

	int tamanio = n * m * l;
	int nm=n * m;

	hipEvent_t ct1, ct2;
	float dt;
	hipEventCreate(&ct1);
	hipEventCreate(&ct2);
	hipEventRecord(ct1);

	kernel<<<grid_size, block_size>>>(r_gpu, g_gpu, b_gpu, tamanio, nm);
	kernel2<<<grid_size, block_size>>>(r_gpu, g_gpu, b_gpu, nm, l);

	hipEventRecord(ct2);
	hipEventSynchronize(ct2);
	hipEventElapsedTime(&dt, ct1, ct2);

	cout << "Tiempo GPU: " << dt << " [ms]" << endl; 

	hipMemcpy(r, r_gpu, sizeof(float) * n * m * l, hipMemcpyDeviceToHost);
	hipMemcpy(g, g_gpu, sizeof(float) * n * m * l, hipMemcpyDeviceToHost);
	hipMemcpy(b, b_gpu, sizeof(float) * n * m * l, hipMemcpyDeviceToHost);

	FILE * pSalida;
	pSalida = fopen ("gpu_img_salida.txt","w");
	fprintf(pSalida, "%d %d\n", m, n);
	for (int i = 0; i < n*m; ++i) {
		if(i == n*m - 1) {
			fprintf(pSalida, "%f", r[i]);
		} else {
			fprintf(pSalida, "%f ", r[i]);
		}
		
	}
	fprintf(pSalida, "\n");
	for (int i = 0; i < n*m; ++i) {
		if(i == n*m - 1) {
			fprintf(pSalida, "%f", g[i]);
		} else {
			fprintf(pSalida, "%f ", g[i]);
		}
	}
	fprintf(pSalida, "\n");
	for (int i = 0; i < n*m; ++i) {
		if(i == n*m - 1) {
			fprintf(pSalida, "%f", b[i]);
		} else {
			fprintf(pSalida, "%f ", b[i]);
		}
	}
	delete r;
	delete g;
	delete b;

	hipFree(r_gpu);
	hipFree(g_gpu);
	hipFree(b_gpu);

	//cin.get();
	return 0;
}